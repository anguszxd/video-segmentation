#include "hip/hip_runtime.h"
#include "opencv2/gpu/device/common.hpp"
#include <opencv2/core/core.hpp>
using namespace cv::gpu;
#include "hip/hip_runtime.h"
#include ""
#include "../Matting/matting.h"
#if HAVE_GPU ==1
#define BLOCK_WIDE 64
#define BLOCK_HIGH 8

#define alpha_top 40
#define alpha_bottom 40
#define alpha_left 120
#define alpha_right 120

namespace cv {
	namespace gpu {
		namespace device {
			/*__device__ const float motion_TH_f = motion_TH / 255.0;
			__device__ const float static_SPEED_f = static_SPEED / 255.0;
			__device__ const float long_SPEED_f = long_SPEED / 255.0;
			__device__ const float luma_offset_f = luma_offset / 255.0f;
			__device__ const float u_gain_f = u_gain;
			__device__ const float v_gain_f = v_gain;*/
			__constant__ TuningParaFloat Const;
			__constant__ HostPara host_para;
			__global__  void trace_bg_kernel(PtrStepSz<float> motion_diff_rgb_filted0, PtrStepSz<float> motion_diff_rgb_filted1, PtrStepSz<float> motion_diff_rgb_filted2, 
					PtrStepSz<float3> frame_yuv, PtrStepSz<float3> bg_yuv, PtrStepSz<float3> bg_diff_yuv, PtrStepSzb static_num, PtrStepSzb is_bg, PtrStepSzb is_body)
			{
				extern __shared__ float smem[];
				typename float * gray = smem;
				unsigned int gray_idx = threadIdx.y * blockDim.x + threadIdx.x;
				unsigned int y = blockIdx.y * (blockDim.y-2) + threadIdx.y;
				unsigned int x = blockIdx.x * (blockDim.x-2) + threadIdx.x;
				
				if (y < static_num.rows && x < static_num.cols) {
					gray[gray_idx] = frame_yuv.ptr(y)[x].x;
					__syncthreads();
					if (threadIdx.x != 0 && threadIdx.y != 0 && threadIdx.x != blockDim.x - 1 && threadIdx.y != blockDim.y - 1 
						&& y + 1<static_num.rows && x + 1<static_num.cols) {
						float edge_offset = MAX(fabs(gray[gray_idx - blockDim.x - 1] - gray[gray_idx + blockDim.x + 1]),
							fabs(gray[gray_idx - blockDim.x + 1] - gray[gray_idx + blockDim.x - 1])) / 2;
						float motion_diff = fabs(motion_diff_rgb_filted0.ptr(y)[x]) + fabs(motion_diff_rgb_filted1.ptr(y)[x]) + fabs(motion_diff_rgb_filted2.ptr(y)[x]);
						unsigned char static_num_reg = static_num.ptr(y)[x];
						if (motion_diff < edge_offset + Const.motion_TH_f)
							static_num_reg = MIN(static_num_reg + 1, Const.static_MAX);
						else
							static_num_reg = 0;
						static_num.ptr(y)[x] = static_num_reg;
						float3 bg_yuv_reg = bg_yuv.ptr(y)[x];
						if (fabs(bg_yuv_reg.x) <= 0.001f && fabs(bg_yuv_reg.y - 1.0f) <= 0.001f && fabs(bg_yuv_reg.z) <=0.001f) {
							if (static_num_reg>= Const.init_static_num) 
								bg_yuv.ptr(y)[x] = frame_yuv.ptr(y)[x];								
						}
						else {
							float update_speed;
							if (is_bg.ptr(y)[x] && static_num_reg >= Const.static_NUM)
								update_speed = Const.static_SPEED_f;
							else if (is_body.ptr(y)[x] == 0 && static_num_reg >= Const.long_static_NUM)
								update_speed = Const.long_SPEED_f;
							else
								update_speed = 0;
							float3 bg_diff_yuv_reg = bg_diff_yuv.ptr(y)[x];
							bg_yuv_reg.x = (bg_diff_yuv_reg.x > 0) ? (bg_yuv_reg.x + update_speed) : (bg_yuv_reg.x - update_speed);
							bg_yuv_reg.y = (bg_diff_yuv_reg.y > 0) ? (bg_yuv_reg.y + update_speed) : (bg_yuv_reg.y - update_speed);
							bg_yuv_reg.z = (bg_diff_yuv_reg.z > 0) ? (bg_yuv_reg.z + update_speed) : (bg_yuv_reg.z - update_speed);
							bg_yuv.ptr(y)[x] = bg_yuv_reg;
						}						
					} 
				}
			}
			
			__global__ void update_mask_bg_kernel(PtrStepSz<float> bg_diff_filted0, PtrStepSz<float> bg_diff_filted1, PtrStepSz<float> bg_diff_filted2, 
					PtrStepSzb fg_sure, PtrStepSzb fg_maybe, PtrStepSzb is_body)
			{			
				unsigned int y = blockIdx.y * blockDim.y + threadIdx.y + alpha_top;
				unsigned int x = blockIdx.x * blockDim.x + threadIdx.x + alpha_left;
				
				float bg_diff_abs_y = fabs(bg_diff_filted0.ptr(y)[x]);
				float bg_diff_abs_u = fabs(bg_diff_filted1.ptr(y)[x]);
				float bg_diff_abs_v = fabs(bg_diff_filted2.ptr(y)[x]);
				
				bg_diff_abs_y = MAX(0.0f, bg_diff_abs_y - Const.luma_offset_f);
				bg_diff_abs_u = bg_diff_abs_u * Const.u_gain_f;
				bg_diff_abs_v = bg_diff_abs_v * Const.v_gain_f;
				float bg_diff_all = (bg_diff_abs_y + bg_diff_abs_u + bg_diff_abs_v)*(fg_sure.ptr(y)[x] + 1);
				float motion_th = Const.alpha_TH_f;
				if ((y >= host_para.body_top - 1) && (y <= host_para.body_bottom - 1) && (x >= host_para.body_left - 1) && (x <= host_para.body_right - 1)) {
					is_body.ptr(y)[x] = 1;
					motion_th = Const.alpha_TH_f / 2;
				} else
					is_body.ptr(y)[x] = 0;

				if (bg_diff_all > motion_th * 2) {					
					fg_sure.ptr(y)[x] = 255;
					fg_maybe.ptr(y)[x] = 255;									
				}
				else {
					fg_sure.ptr(y)[x] = 0;					
					if (bg_diff_all > motion_th)
						fg_maybe.ptr(y)[x] = 255;
					else 
						fg_maybe.ptr(y)[x] = 0;				
				}
			}
			
			void trace_bg_(PtrStepSzb motion_diff_rgb_filted0, PtrStepSzb motion_diff_rgb_filted1, PtrStepSzb motion_diff_rgb_filted2, 
					PtrStepSzb frame_yuv, PtrStepSzb bg_yuv, PtrStepSzb bg_diff_yuv, PtrStepSzb static_num, PtrStepSzb is_bg, PtrStepSzb is_body, hipStream_t stream)
			{
				const dim3 block(BLOCK_WIDE, BLOCK_HIGH);
				const dim3 grid(divUp(frame_yuv.cols - 2, BLOCK_WIDE - 2), divUp(frame_yuv.rows - 2, BLOCK_HIGH - 2));
				const size_t smemSize = BLOCK_WIDE * BLOCK_HIGH * sizeof(float);
				
				trace_bg_kernel<< <grid, block, smemSize, stream >> > (static_cast<PtrStepSz<float>>(motion_diff_rgb_filted0), static_cast<PtrStepSz<float>>(motion_diff_rgb_filted1), static_cast<PtrStepSz<float>>(motion_diff_rgb_filted2), 
					static_cast<PtrStepSz<float3>>(frame_yuv), static_cast<PtrStepSz<float3>>(bg_yuv), static_cast<PtrStepSz<float3>>(bg_diff_yuv), static_num, is_bg, is_body);
			}
			
			void update_mask_bg_(PtrStepSzb bg_diff_filted0, PtrStepSzb bg_diff_filted1, PtrStepSzb bg_diff_filted2, 
					PtrStepSzb fg_sure, PtrStepSzb fg_maybe, PtrStepSzb is_body, hipStream_t stream)
			{
				const dim3 block(BLOCK_WIDE, BLOCK_HIGH);
				const dim3 grid(divUp(fg_sure.cols - alpha_left - alpha_right, BLOCK_WIDE), divUp(fg_sure.rows - alpha_top - alpha_bottom, BLOCK_HIGH));
				const size_t smemSize = 0;
				
				update_mask_bg_kernel << <grid, block, smemSize, stream >> > (static_cast<PtrStepSz<float>>(bg_diff_filted0), static_cast<PtrStepSz<float>>(bg_diff_filted1), static_cast<PtrStepSz<float>>(bg_diff_filted2),
					fg_sure, fg_maybe, is_body);
			}
		}
	}
}

void trace_bg(PtrStepSzb motion_diff_rgb_filted0, PtrStepSzb motion_diff_rgb_filted1, PtrStepSzb motion_diff_rgb_filted2, 
					PtrStepSzb frame_yuv, PtrStepSzb bg_yuv, PtrStepSzb bg_diff_yuv, PtrStepSzb static_num, PtrStepSzb is_bg, PtrStepSzb is_body, hipStream_t stream)
{
	CV_Assert(motion_diff_rgb_filted0.cols==is_bg.cols && frame_yuv.cols==is_bg.cols && bg_yuv.cols==is_bg.cols && bg_diff_yuv.cols==is_bg.cols
		&& static_num.cols==is_bg.cols && is_body.cols==is_bg.cols);
	CV_Assert(motion_diff_rgb_filted0.rows==is_bg.rows && frame_yuv.rows==is_bg.rows && bg_yuv.rows==is_bg.rows && bg_diff_yuv.rows==is_bg.rows
		&& static_num.rows==is_bg.rows && is_body.rows==is_bg.rows);
		
	device::trace_bg_(motion_diff_rgb_filted0, motion_diff_rgb_filted1, motion_diff_rgb_filted2, frame_yuv, bg_yuv, 
		bg_diff_yuv, static_num, is_bg, is_body, stream);
}

void update_mask_bg(PtrStepSzb bg_diff_filted0, PtrStepSzb bg_diff_filted1, PtrStepSzb bg_diff_filted2, 
					PtrStepSzb fg_sure, PtrStepSzb fg_maybe, PtrStepSzb is_body, hipStream_t stream)
{
	CV_Assert(bg_diff_filted0.cols==is_body.cols && bg_diff_filted1.cols==is_body.cols && bg_diff_filted2.cols==is_body.cols
		&& fg_sure.cols==is_body.cols && fg_maybe.cols==is_body.cols);
	CV_Assert(bg_diff_filted0.rows==is_body.rows && bg_diff_filted1.rows==is_body.rows && bg_diff_filted2.rows==is_body.rows
		&& fg_sure.rows==is_body.rows && fg_maybe.rows==is_body.rows);

	device::update_mask_bg_(bg_diff_filted0, bg_diff_filted1, bg_diff_filted2, fg_sure, fg_maybe, is_body, stream);
}

void tune_gpu_parameter(TuningParaFloat *c)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(device::Const), c, sizeof(TuningParaFloat)));
}

void update_host_para(HostPara *p)
{
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(device::host_para), p, sizeof(HostPara)));
}

#endif